// stdlib
#include <cassert>
#include <cinttypes>
#include <cstdio>
#include <cstring>
#include <fstream>
#include <vector>

// CUDA related
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// own
#include "sake.hpp"
#include "sha256.cuh"

#define CUDA_DRV_CHECK(e) do {\
    hipError_t r = e;\
    if (r != hipSuccess) {\
       const char* ptr = nullptr;\
       hipDrvGetErrorString(e, &ptr);\
       if (ptr) {\
           printf("CUDA ERROR %s:%d %s %s\n", __FILE__, __LINE__, #e, ptr);\
       } else {\
           printf("CUDA ERROR %s:%d %s INVALID ERROR CODE\n", __FILE__, __LINE__, #e);\
       }\
       exit(1);\
    }\
} while (0)

void sake_runner() {
    printf("[G] Running SAKE protocol...\n");
}